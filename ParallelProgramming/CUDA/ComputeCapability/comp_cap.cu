#include <stdio.h>
#include <stdlib.h>

void main(void)
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	printf("Compute power: %d.%d\n", deviceProp.major, deviceProp.minor);
	hipDeviceReset();
}