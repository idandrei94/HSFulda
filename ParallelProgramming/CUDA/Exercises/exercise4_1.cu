
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#define P 2
#define Q 3
#define R 4

__device__ __constant__ double* cuda_matrix_b;

/* define macro to check the return value of a CUDA function */
#define CheckRetValueOfCudaFunction(val) \
 if (val != hipSuccess) \
 { \
 fprintf (stderr, "file: %s line %d: %s.\n", \
 __FILE__, __LINE__, hipGetErrorString (val)); \
 hipDeviceReset (); \
 exit (EXIT_FAILURE); \
 }

__global__ void printKernel( const double * __restrict__ arr, const size_t q, const size_t r)
{
  printf("%d x %d\n", q, r);
  int tid = (int) (blockIdx.x * blockDim.x + threadIdx.x);
  if(tid < q)
  {
    for(int j = 0; j < r; ++j)
    {
      printf("(tid %d) Printing element[%d][%d]: %f\n", tid, tid, j, arr[tid*q+j]);
    }
    printf("\n");
  }
}

void init_arrays(double **a, double *b, double **c)
{
  for(int i = 0; i < P; ++i)
  {
    for(int j = 0; j < Q; ++j)
    {
      a[i][j] = i * P + j;
    }
  }

  for(int i = 0; i < Q; ++i)
  {
    for(int j = 0; j < R; ++j)
    {
      b[i*Q+j] = i * Q + j;
    }
  }

  for(int i = 0; i < P; ++i)
  {
    for(int j = 0; j < R; ++j)
    {
      c[i][j] = 0;
    }
  }
}

int main( void )
{
  hipError_t cuda_ret;		
	time_t start_time;
	clock_t start_clock;

  double **a;
  double *b;
  double **c;

  start_clock = clock();
  start_time = time(NULL);


  // MEMORY ALLOCATION
  a = (double**)malloc(sizeof(double*) * P);

  b = (double*)malloc(sizeof(double) * Q * R);
  cuda_ret = hipMalloc((void**)&cuda_matrix_b, sizeof(double) * Q * R);
  CheckRetValueOfCudaFunction(cuda_ret);

  c = (double**)malloc(sizeof(double*)*P);

  for(int i = 0; i < P; ++i)
  {
    a[i] = (double*)malloc(sizeof(double) * Q);
    c[i] = (double*)malloc(sizeof(double) * R);
  }

  // INIT LOCAL DATA
  init_arrays(a, b, c);
  hipMemcpy(cuda_matrix_b, b, sizeof(double) * Q * R, hipMemcpyHostToDevice);

  // DO THE MAGIC

  printKernel <<<5, 6>>> (cuda_matrix_b , Q, R);
  cuda_ret = hipDeviceSynchronize ();
  CheckRetValueOfCudaFunction (cuda_ret);


  // FREE 
  for(int i = 0; i < P; ++i)
  {
    free(c[i]);
  }
  
  for(int i = 0; i < Q; ++i)
  {
    free(a[i]);
    CheckRetValueOfCudaFunction(cuda_ret);
  }

  free(a);
  free(b);
  free(c);

  hipFree(&cuda_matrix_b);
  CheckRetValueOfCudaFunction(cuda_ret);

  hipDeviceReset();
  CheckRetValueOfCudaFunction(cuda_ret);

  printf("CPU time:\t\t%.2f\t\tReal time: \t\t%.2f",
		((double)(clock()-start_clock))/CLOCKS_PER_SEC, 
		difftime(time(NULL), start_time));
  return EXIT_SUCCESS;
}
